#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include "evotunf_ext.h"

// #ifndef NDEBUG
#define CUDA_CALL(x) do { hipError_t e; if ((e = (x)) != hipSuccess) { \
    fprintf(stderr, "CUDA Error at %s:%d %s { %s }\n", __FILE__, __LINE__, hipGetErrorString(e), #x); \
    switch (e) { \
        case hipErrorInvalidValue: fprintf(stderr, "InvalidValue\n"); break; \
        case hipErrorOutOfMemory: fprintf(stderr, "MemoryAllocation\n"); break; \
        case hipErrorHostMemoryAlreadyRegistered: fprintf(stderr, "HostMemoryAlreadyRegistered\n"); break; \
        case hipErrorNotSupported: fprintf(stderr, "NotSupported\n"); break; \
    } \
    exit(EXIT_FAILURE); } } while (0)
// #else
// #define CUDA_CALL(x) (x)
// #endif

#define LUKASZEWICZ_IMPL(a, b) fmin(1.f, 1.f - (a) + (b))
#define IMPL(a, b) LUKASZEWICZ_IMPL(a, b)
#define GAUSS(mu, sigma, x) expf(-powf(((x) - (mu)) / (sigma), 2))

typedef union {
    float cross;
    struct {
        float numerator;
        float denominator;
    };
} FractionEx;

__device__ float evaluate_kernel(
        const unsigned *fsets_offsets, const GaussParams *gauss_params,
        const unsigned char *rules, const GaussParams *xx)
{
    unsigned step, j;
    unsigned rules_len = blockDim.y;
    unsigned n = blockDim.x;
    unsigned rule_idx = threadIdx.y;
    unsigned attr_idx = threadIdx.x;

    extern __shared__ char cache[];
    FractionEx *res_cache = (FractionEx*)cache;
    float *impl_cache = (float*)(cache + rules_len * sizeof(FractionEx));

    GaussParams ux = xx[attr_idx];

    {
        float y_center = gauss_params[fsets_offsets[n] + rules[rule_idx * (n+1) + n]].mu;
        float cross = 1.f;

        for (j = 0; j < rules_len; ++j) {
            GaussParams ua = gauss_params[fsets_offsets[attr_idx] + rules[j * (n+1) + attr_idx]];
            GaussParams ub = gauss_params[fsets_offsets[n] + rules[j * (n+1) + n]];
            float x, sup = 0.f;

            for (x = 0.f; x <= 1.f; x += 0.1f) {
                float impl = LUKASZEWICZ_IMPL(GAUSS(ua.mu, ua.sigma, x), GAUSS(ub.mu, ub.sigma, y_center));
                float t_norm = fmin(GAUSS(ux.mu, ux.sigma, x), impl);
                if (t_norm > sup) sup = t_norm;
            }
            impl_cache[j * n + attr_idx] = sup;

            float *impl_cache_row = impl_cache + j * n;
            for (step = 1; step < n; step <<= 1) {
                if (!(attr_idx & ((step<<1)-1)) && attr_idx + step < n) {
                    if (impl_cache_row[attr_idx + step] > impl_cache_row[attr_idx]) {
                        impl_cache_row[attr_idx] = impl_cache_row[attr_idx + step];
                    }
                }
            }
            if (threadIdx.x == 0) {
                if (impl_cache[j * n] < cross) cross = impl_cache[j * n];
            }
        }

        if (threadIdx.x == 0) {
            res_cache[rule_idx].numerator = y_center * cross;
            res_cache[rule_idx].denominator = cross;
        }
    }

    if (threadIdx.x == 0) {
        for (step = 1; step < rules_len; step <<= 1) {
            if (!(rule_idx & ((step<<1)-1)) && rule_idx + step < rules_len) {
                res_cache[rule_idx].numerator += res_cache[rule_idx + step].numerator;
                res_cache[rule_idx].denominator += res_cache[rule_idx + step].denominator;
            }
        }
    }

    return (res_cache[0].denominator) ? res_cache[0].numerator / res_cache[0].denominator : 0.f;
}


__global__ void predict_kernel(
        // enum t_norm t_outer, enum t_norm t_inner, enum impl impl,
        const unsigned *fsets_offsets, const GaussParams *gauss_params, const unsigned char *rules,
        const GaussParams *xxs, size_t xxs_pitch, unsigned *ys)
{
    unsigned idx = blockIdx.x;
    ys[idx] = evaluate_kernel(fsets_offsets, gauss_params, rules, (GaussParams*)((char*)xxs + idx * xxs_pitch));
}


extern "C"
void predict_gpu_impl(
        // enum t_norm t_outer, enum t_norm t_inner, enum impl impl,
        const unsigned *fsets_lens, const GaussParams *gauss_params, const unsigned char *rules, unsigned rules_len, unsigned n,
        const GaussParams *xxs, unsigned *ys, unsigned N)
{
    /*
     * GPU memory layout
     * +--------------------------
     * | fsets_offsets
     * +--------------------------
     * | GaussParams array [input * fsets_len[input]]
     * +--------------------------
     * | Rules array
     * +--------------------------
     * | Input array
     * +--------------------------
     * | Output buffer
     * +--------------------------
     */

    unsigned i, fsets_total_len = 0, offset = 0;
    unsigned fsets_offsets[n+1];

    for (i = 0; i < n+1; ++i) {
        fsets_offsets[i] = offset;
        offset += fsets_lens[i];
        fsets_total_len += fsets_lens[i];
    }

    size_t xxs_d_pitch;
    unsigned *fsets_offsets_d;
    GaussParams *gauss_params_d, *xxs_d;
    unsigned char *rules_d;
    unsigned *ys_d;

    CUDA_CALL(hipMalloc(&fsets_offsets_d, sizeof(unsigned[n+1])));
    CUDA_CALL(hipMalloc(&gauss_params_d, sizeof(GaussParams[fsets_total_len])));
    CUDA_CALL(hipMalloc(&rules_d, sizeof(unsigned char[rules_len][n+1])));
    CUDA_CALL(hipMallocPitch(&xxs_d, &xxs_d_pitch, sizeof(GaussParams[n]), N));
    CUDA_CALL(hipMalloc(&ys_d, sizeof(unsigned[N])));

    hipMemcpy(fsets_offsets_d, fsets_offsets, sizeof(unsigned[n+1]), hipMemcpyHostToDevice);
    hipMemcpy(gauss_params_d, gauss_params, sizeof(GaussParams[fsets_total_len]), hipMemcpyHostToDevice);
    hipMemcpy(rules_d, rules, sizeof(unsigned char[rules_len][n+1]), hipMemcpyHostToDevice);
    hipMemcpy2D(xxs_d, xxs_d_pitch, xxs, sizeof(GaussParams[n]), sizeof(GaussParams[n]), N, hipMemcpyHostToDevice);

    {
        size_t shared_sz = sizeof(FractionEx[rules_len]) + sizeof(float[rules_len][n]);
        predict_kernel<<<N, dim3(n, rules_len), shared_sz>>>(fsets_offsets_d, gauss_params_d, rules_d, xxs_d, xxs_d_pitch, ys_d);
    }

    hipMemcpy(ys, ys_d, sizeof(unsigned[N]), hipMemcpyDeviceToHost);

    hipFree(ys_d);
    hipFree(xxs_d);
    hipFree(rules_d);
    hipFree(gauss_params_d);
    hipFree(fsets_offsets_d);
}

typedef hiprandStatePhilox4_32_10_t RandomState;

__global__ void initialize_random_states_kernel(RandomState *states, size_t states_pitch)
{
    unsigned chromosome_idx = blockIdx.x;
    unsigned param_idx = threadIdx.x;
    unsigned state_idx = chromosome_idx * blockDim.x + param_idx;

    RandomState *state = (RandomState*)((char*)states + chromosome_idx * states_pitch) + param_idx;
    hiprand_init(1234, state_idx, 0, state);
}

static
void initialize_random_states(
        RandomState *params_random_states_d, size_t params_random_states_d_pitch, RandomState *rules_random_states_d, size_t rules_random_states_d_pitch,
        unsigned new_population_power, unsigned fsets_total_len, unsigned rules_len)
{
    initialize_random_states_kernel<<<new_population_power, fsets_total_len>>>(params_random_states_d, params_random_states_d_pitch);
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipPeekAtLastError());
    initialize_random_states_kernel<<<new_population_power, rules_len>>>(rules_random_states_d, rules_random_states_d_pitch);
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipPeekAtLastError());
}

__global__ void initialize_params_kernel(
        RandomState *states, size_t states_pitch, const DataBounds *data_bounds_expanded, unsigned *fsets_lens_expanded,
        GaussParams *gauss_params, size_t gauss_params_pitch, EvolutionaryParams *evolutionary_params, size_t evolutionary_params_pitch, float h)
{
    unsigned chromosome_idx = blockIdx.x;
    unsigned param_idx = threadIdx.x;

    DataBounds db = data_bounds_expanded[param_idx];
    unsigned fsets_len = fsets_lens_expanded[param_idx];

    GaussParams *gp = (GaussParams*)((char*)gauss_params + chromosome_idx * gauss_params_pitch) + param_idx;
    RandomState *state = (RandomState*)((char*)states + chromosome_idx * states_pitch) + param_idx;
    gp->mu = (hiprand(state) % fsets_len + 0.5f) / fsets_len;
    gp->sigma = 0.5f / fsets_len;

    EvolutionaryParams *ep = (EvolutionaryParams*)((char*)evolutionary_params + chromosome_idx * evolutionary_params_pitch) + param_idx;
    ep->sigma1 = ep->sigma2 = h / fsets_len;
}

__global__ void initialize_rules_kernel(
        RandomState *states, size_t states_pitch, const unsigned *fsets_lens,
        unsigned char *rules, size_t rules_pitch, unsigned n)
{
    unsigned i;
    unsigned chromosome_idx = blockIdx.x;
    unsigned rule_idx = threadIdx.x;
    size_t state_offset = chromosome_idx * states_pitch + rule_idx;

    RandomState local_state = ((RandomState*)((char*)states + chromosome_idx * states_pitch))[rule_idx];

    for (i = 0; i < n+1; ++i) {
        rules[chromosome_idx * rules_pitch + rule_idx * (n+1) + i] = hiprand(&local_state) % fsets_lens[i];
    }

    ((RandomState*)((char*)states + chromosome_idx * states_pitch))[rule_idx] = local_state;
}

static
void initialize_population(
        RandomState *params_random_states_d, size_t params_random_states_d_pitch, RandomState *rules_random_states_d, size_t rules_random_states_d_pitch,
        const DataBounds *data_bounds, const unsigned *fsets_lens, const unsigned *fsets_lens_d, unsigned fsets_total_len, unsigned population_power,
        GaussParams *gauss_params_d, size_t gauss_params_d_pitch, EvolutionaryParams *evolutionary_params_d, size_t evolutionary_params_d_pitch,
        unsigned char *rules_d, size_t rules_d_pitch, unsigned rules_len, unsigned n, float h)
{
    /*
     * Local GPU memory layout:
     * +==============================
     * |      ...................
     * | NEW rules array [new_population_power, rules_len, n+1]
     * +==============================
     * | DataBounds buffer [n+1]
     * +------------------------------
     * | fsets_lens_expanded buffer [fsets_total_len]
     * +==============================
     */

    size_t i, k, offset = 0;
    DataBounds data_bounds_expanded[fsets_total_len];
    unsigned fsets_lens_expanded[fsets_total_len];

    for (i = 0; i < n+1; ++i) {
        unsigned fsets_len = fsets_lens[i];
        float a = (data_bounds[i].max - data_bounds[i].min) / fsets_len;
        for (k = 0; k < fsets_len; ++k) {
            fsets_lens_expanded[offset + k] = fsets_len;
            data_bounds_expanded[offset + k].min = data_bounds[i].min;
            data_bounds_expanded[offset + k].a = a;
        }
        offset += fsets_len;
    }

    DataBounds *data_bounds_expanded_d;
    unsigned *fsets_lens_expanded_d;

    CUDA_CALL(hipMalloc(&data_bounds_expanded_d, sizeof(DataBounds[fsets_total_len])));
    CUDA_CALL(hipMalloc(&fsets_lens_expanded_d, sizeof(unsigned[fsets_total_len])));

    CUDA_CALL(hipMemcpy(data_bounds_expanded_d, data_bounds_expanded, sizeof(DataBounds[fsets_total_len]), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(fsets_lens_expanded_d, fsets_lens_expanded, sizeof(unsigned[fsets_total_len]), hipMemcpyHostToDevice));

    initialize_params_kernel<<<population_power, fsets_total_len>>>(
            params_random_states_d, params_random_states_d_pitch, data_bounds_expanded_d, fsets_lens_expanded_d,
            gauss_params_d, gauss_params_d_pitch, evolutionary_params_d, evolutionary_params_d_pitch, h);
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipPeekAtLastError());

    initialize_rules_kernel<<<population_power, rules_len>>>(
            rules_random_states_d, rules_random_states_d_pitch, fsets_lens_d,
            rules_d, rules_d_pitch, n);
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipPeekAtLastError());

    CUDA_CALL(hipFree(fsets_lens_expanded_d));
    CUDA_CALL(hipFree(data_bounds_expanded_d));
}

#define BLOCK_SIZE 64

__global__ void accumulate_scores_kernel(
        const unsigned *fsets_offsets, const GaussParams *gauss_params, size_t gauss_params_pitch,
        const unsigned char *rules, size_t rules_pitch,
        const GaussParams *xxs, size_t xxs_pitch, const unsigned *ys, float *scores)
{
    unsigned chromosome_idx = blockIdx.y;
    unsigned data_idx = blockIdx.x;

    float pred = evaluate_kernel(
            fsets_offsets, (GaussParams*)((char*)gauss_params + chromosome_idx * gauss_params_pitch),
            rules + chromosome_idx * rules_pitch, (GaussParams*)((char*)xxs + data_idx * xxs_pitch));
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        atomicAdd(scores + chromosome_idx, powf(ys[data_idx] - pred, 2));
    }
}

__global__ void normalize_scores_kernel(float *scores, unsigned population_power, unsigned N)
{
    unsigned tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (tid < population_power) scores[tid] = -sqrtf(scores[tid] / N);
}

void compute_scores(
        unsigned population_power,
        const unsigned *fsets_offsets_d, const GaussParams *gauss_params_d, size_t gauss_params_d_pitch,
        const unsigned char *rules_d, size_t rules_d_pitch, unsigned rules_len, unsigned n,
        const GaussParams *xxs_d, size_t xxs_d_pitch,
        const unsigned *ys_d, unsigned N, float *scores_d)
{
    hipMemset(scores_d, 0, sizeof(float[population_power]));

    {
        size_t shared_sz = sizeof(FractionEx[rules_len]) + sizeof(float[rules_len][n]);
        accumulate_scores_kernel<<<dim3(N, population_power), dim3(n, rules_len), shared_sz>>>(
                fsets_offsets_d, gauss_params_d, gauss_params_d_pitch,
                rules_d, rules_d_pitch, xxs_d, xxs_d_pitch, ys_d, scores_d);
    }
    CUDA_CALL(hipDeviceSynchronize());
    normalize_scores_kernel<<<(population_power + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(scores_d, population_power, N);
    CUDA_CALL(hipDeviceSynchronize());
}

__global__ void generate_random_indices_kernel(RandomState *states, size_t states_pitch, unsigned *indices, unsigned population_power, unsigned k)
{
    unsigned idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (idx < population_power) {
        RandomState *state = (RandomState*)((char*)states + blockIdx.x * states_pitch) + threadIdx.x;
        indices[idx] = hiprand(state) % k;
    }
}

__global__ void copy_params_for_given_chromosomes_kernel(
        const unsigned *chromosome_indices,
        GaussParams *new_gauss_params, const GaussParams *gauss_params, size_t gauss_params_pitch,
        EvolutionaryParams *new_evolutionary_params, const EvolutionaryParams *evolutionary_params, size_t evolutionary_params_pitch)
{
    unsigned new_chromosome_idx = blockIdx.x;
    unsigned param_idx = threadIdx.x;

    unsigned chromosome_idx = chromosome_indices[new_chromosome_idx];

    GaussParams *ngps = (GaussParams*)((char*)new_gauss_params + new_chromosome_idx * gauss_params_pitch);
    GaussParams *gps = (GaussParams*)((char*)gauss_params + chromosome_idx * gauss_params_pitch);
    ngps[param_idx] = gps[param_idx];

    EvolutionaryParams *neps = (EvolutionaryParams*)((char*)new_evolutionary_params + new_chromosome_idx * evolutionary_params_pitch);
    EvolutionaryParams *eps = (EvolutionaryParams*)((char*)evolutionary_params + chromosome_idx * evolutionary_params_pitch);
    neps[param_idx] = eps[param_idx];
}

__global__ void copy_rules_for_given_chromosomes_kernel(
        const unsigned *chromosome_indices,
        unsigned char *new_rules, const unsigned char *rules, size_t rules_pitch,
        unsigned n)
{
    unsigned i;
    unsigned new_chromosome_idx = blockIdx.x;
    unsigned rules_len = blockDim.x;
    unsigned rule_idx = threadIdx.x;

    unsigned chromosome_idx = chromosome_indices[new_chromosome_idx];
    for (i = 0; i < n+1; ++i) {
        new_rules[new_chromosome_idx * rules_pitch + rule_idx * (n+1) + i] = rules[chromosome_idx * rules_pitch + rule_idx * (n+1) + i];
    }
}

static
void perform_reproduction(
        RandomState *random_states_d, size_t random_states_d_pitch,
        unsigned *chromosome_indices_d, unsigned population_power, unsigned new_population_power,
        GaussParams *new_gauss_params_d, const GaussParams *gauss_params_d, size_t gauss_params_d_pitch,
        EvolutionaryParams *new_evolutionary_params_d, const EvolutionaryParams *evolutionary_params_d, size_t evolutionary_params_d_pitch,
        unsigned char *new_rules_d, const unsigned char *rules_d, size_t rules_d_pitch, unsigned fsets_total_len, unsigned rules_len, unsigned n)
{
    generate_random_indices_kernel<<<(new_population_power + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
            random_states_d, random_states_d_pitch, chromosome_indices_d, new_population_power, population_power);
    copy_params_for_given_chromosomes_kernel<<<new_population_power, fsets_total_len>>>(
            chromosome_indices_d, new_gauss_params_d, gauss_params_d, gauss_params_d_pitch,
            new_evolutionary_params_d, evolutionary_params_d, evolutionary_params_d_pitch);
    copy_rules_for_given_chromosomes_kernel<<<new_population_power, rules_len>>>(
            chromosome_indices_d, new_rules_d, rules_d, rules_d_pitch, n);
}

__global__ void find_max_score(float *scores, unsigned *indices, unsigned total_len, unsigned offset)
{
    unsigned step;
    unsigned chromosome_idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    extern __shared__ char cache[];
    float *scores_cache = (float*)cache;
    unsigned *indices_cache = (unsigned*)(cache + BLOCK_SIZE * sizeof(float));

    if (chromosome_idx < total_len) {
        scores_cache[threadIdx.x] = scores[offset + chromosome_idx];
        indices_cache[threadIdx.x] = offset + chromosome_idx;
    }

    for (step = 1; step < BLOCK_SIZE; step <<= 1) {
        if ((threadIdx.x & ((step<<1)-1)) == 0 && chromosome_idx + step < total_len) {
            if (!isnan(scores_cache[threadIdx.x + step]) && scores_cache[threadIdx.x + step] > scores_cache[threadIdx.x]
                    || isnan(scores_cache[threadIdx.x])) {
                scores_cache[threadIdx.x] = scores_cache[threadIdx.x + step];
                indices_cache[threadIdx.x] = indices_cache[threadIdx.x + step];
            }
        }
    }

    if (threadIdx.x == 0) {
        indices[offset + blockIdx.x] = indices_cache[0];
    }
}

__global__ void find_max_score_from_indices(float *scores, unsigned *indices, unsigned offset)
{
    unsigned step;
    unsigned idx = threadIdx.x;
    unsigned dim = blockDim.x;

    extern __shared__ char cache[];
    float *scores_cache = (float*)cache;
    unsigned *indices_cache = (unsigned*)(cache + dim * sizeof(float));

    {
        unsigned score_idx = indices_cache[idx] = indices[offset + idx];
        scores_cache[idx] = scores[score_idx];
    }
    for (step = 1; step < dim; step <<= 1) {
        if ((idx & ((step<<1)-1)) == 0 && idx + step < dim) {
            if (scores_cache[idx + step] > scores_cache[idx]) {
                scores_cache[idx] = scores_cache[idx + step];
                indices_cache[idx] = indices_cache[idx + step];
            }
        }
    }

    if (threadIdx.x == 0) {
        scores[indices_cache[0]] = -INFINITY;
        scores[offset] = scores_cache[0];
        indices[offset] = indices_cache[0];
    }
}

static
void perform_selection(
        float *chromosome_scores_d, unsigned *chromosome_indices_d, unsigned population_power, unsigned new_population_power,
        GaussParams *gauss_params_d, const GaussParams *new_gauss_params_d, size_t gauss_params_d_pitch,
        EvolutionaryParams *evolutionary_params_d, const EvolutionaryParams *new_evolutionary_params_d, size_t evolutionary_params_d_pitch,
        unsigned char *rules_d, const unsigned char *new_rules_d, size_t rules_d_pitch, unsigned fsets_total_len, unsigned rules_len, unsigned n)
{
    unsigned i;

    for (i = 0; i < population_power; ++i) {
        unsigned len = new_population_power - i;
        unsigned dim = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
        {
            size_t shared_sz = sizeof(float[len]) + sizeof(unsigned[len]);
            find_max_score<<<dim, BLOCK_SIZE, shared_sz>>>(chromosome_scores_d, chromosome_indices_d, len, i);
            CUDA_CALL(hipDeviceSynchronize());
        }
        {
            size_t shared_sz = sizeof(float[dim]) + sizeof(unsigned[dim]);
            find_max_score_from_indices<<<1, dim, shared_sz>>>(chromosome_scores_d, chromosome_indices_d, i);
            CUDA_CALL(hipDeviceSynchronize());
        }
    }

    copy_params_for_given_chromosomes_kernel<<<population_power, fsets_total_len>>>(
            chromosome_indices_d, gauss_params_d, new_gauss_params_d, gauss_params_d_pitch,
            evolutionary_params_d, new_evolutionary_params_d, evolutionary_params_d_pitch);
    copy_rules_for_given_chromosomes_kernel<<<population_power, rules_len>>>(
            chromosome_indices_d, rules_d, new_rules_d, rules_d_pitch, n);
}

// static void perform_reproduction_and_selection(
//         float *chromosome_scores_d, unsigned *chromosome_indices_d, unsigned population_power, unsigned new_population_power,
//         GaussParams *gauss_params_d, size_t gauss_params_d_pitch,
//         EvolutionaryParams *evolutionary_params_d, size_t evolutionary_params_d_pitch,
//         unsigned char *rules_d, size_t rules_d_pitch, unsigned fsets_total_len, unsigned rules_len, unsigned n)
// {

// }

__global__ void generate_random_normal_numbers_kernel(RandomState *states, size_t states_pitch, unsigned population_power, float *numbers)
{
    unsigned idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (idx < population_power) {
        RandomState *state = (RandomState*)((char*)states + idx * states_pitch);
        numbers[idx] = hiprand_normal(state);
    }
}

__global__ void perform_params_mutation_kernel(
        RandomState *states, size_t states_pitch, const float *ps,
        GaussParams *gauss_params, size_t gauss_params_pitch, EvolutionaryParams *evolutionary_params, size_t evolutionary_params_pitch)
{
    unsigned chromosome_idx = blockIdx.x;
    unsigned fsets_total_len = blockDim.x;
    unsigned param_idx = threadIdx.x;

    unsigned chromosome_len = fsets_total_len * (/* gauss.mu */ 1 + /* gauss.sigma */ 1);
    float tau1 = 1.f / sqrtf(2.f * chromosome_len);
    float tau = 1.f / sqrtf(2.f * sqrtf(chromosome_len));

    RandomState local_state = ((RandomState*)((char*)states + chromosome_idx * states_pitch))[param_idx];
    float p = ps[chromosome_idx];

    EvolutionaryParams *ep = (EvolutionaryParams*)((char*)evolutionary_params + chromosome_idx * evolutionary_params_pitch) + param_idx;
    float sigma1 = ep->sigma1 *= expf(tau1 * p + tau * hiprand_normal(&local_state));
    float sigma2 = ep->sigma2 *= expf(tau1 * p + tau * hiprand_normal(&local_state));

    GaussParams *gp = (GaussParams*)((char*)gauss_params + chromosome_idx * gauss_params_pitch) + param_idx;
    gp->mu += sigma1 * hiprand_normal(&local_state);
    gp->sigma += sigma2 * hiprand_normal(&local_state);

    ((RandomState*)((char*)states + chromosome_idx * states_pitch))[param_idx] = local_state;
}

__global__ void perform_rules_mutation_kernel(
        RandomState *states, size_t states_pitch, const unsigned *fsets_lens,
        unsigned char *rules, size_t rules_pitch, unsigned n, float pm)
{
    unsigned i;
    unsigned chromosome_idx = blockIdx.x;
    unsigned rule_idx = threadIdx.x;

    unsigned char *rule_row = rules + chromosome_idx * rules_pitch + rule_idx * (n+1);
    RandomState local_state = ((RandomState*)((char*)states + chromosome_idx * states_pitch))[rule_idx];
    for (i = 0; i < n+1; ++i) {
        if (hiprand_uniform(&local_state) <= pm) {
            rule_row[i] = hiprand(&local_state) % fsets_lens[i];
            // printf("%u %u %u: %u\n", chromosome_idx, rule_idx, i, rule_row[i]);
        }
        __syncthreads();
    }
    ((RandomState*)((char*)states + chromosome_idx * states_pitch))[rule_idx] = local_state;
}

static
void perform_mutation(
        RandomState *params_random_states_d, size_t params_random_states_d_pitch, RandomState *rules_random_states_d, size_t rules_random_states_d_pitch,
        float *ps_d, unsigned new_population_power, const unsigned *fsets_lens_d, unsigned fsets_total_len,
        GaussParams *gauss_params_d, size_t gauss_params_d_pitch, EvolutionaryParams *evolutionary_params_d, size_t evolutionary_params_d_pitch,
        unsigned char *rules_d, size_t rules_d_pitch, unsigned rules_len, unsigned n, float pm)
{
    generate_random_normal_numbers_kernel<<<(new_population_power + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
            params_random_states_d, params_random_states_d_pitch, new_population_power, ps_d);
    CUDA_CALL(hipDeviceSynchronize());
    perform_params_mutation_kernel<<<new_population_power, fsets_total_len>>>(
            params_random_states_d, params_random_states_d_pitch, ps_d,
            gauss_params_d, gauss_params_d_pitch, evolutionary_params_d, evolutionary_params_d_pitch);
    CUDA_CALL(hipDeviceSynchronize());
    perform_rules_mutation_kernel<<<new_population_power, rules_len>>>(
            rules_random_states_d, rules_random_states_d_pitch, fsets_lens_d,
            rules_d, rules_d_pitch, n, pm);
    CUDA_CALL(hipDeviceSynchronize());
}

__global__
void generate_random_normal_indices_kernel(RandomState *states, size_t states_pitch, unsigned population_power, unsigned rules_len, unsigned n, unsigned *indices)
{
    unsigned chromosome_idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (chromosome_idx < population_power) {
        RandomState *state = (RandomState*)((char*)states + chromosome_idx * states_pitch);
        indices[chromosome_idx] = (unsigned)(hiprand_normal(state) * (rules_len * (n+1)));
    }
}

__global__
void perform_crossingover_kernel(const unsigned *rules_offsets, unsigned char *rules, size_t rules_pitch, unsigned rules_len, unsigned n)
{
    unsigned chromosome_pair_idx = blockIdx.x;
    unsigned i;

    unsigned rules_offset = rules_offsets[chromosome_pair_idx];
    unsigned char *rules_a = rules + 2 * chromosome_pair_idx;
    unsigned char *rules_b = rules + 2 * chromosome_pair_idx + 1;

    extern __shared__ unsigned char rules_cache[];
    unsigned char *rules_a_cache = rules_cache;
    unsigned char *rules_b_cache = rules_cache + rules_len * (n+1);

    for (i = threadIdx.x + rules_offset / blockDim.x; i < rules_len * (n+1); ++i) {
        rules_a_cache[i] = rules_a[i];
        rules_b_cache[i] = rules_b[i];
    }

    for (i = threadIdx.x + rules_offset / blockDim.x; i < rules_len * (n+1); ++i) {
        if (i >= rules_offset) {
            rules_a[i] = rules_b_cache[i];
            rules_b[i] = rules_a_cache[i];
        }
    }
}

static
void perform_crossingover(
        RandomState *rules_random_states_d, size_t rules_random_states_d_pitch, unsigned *indices_d, unsigned population_power,
        unsigned char *rules_d, size_t rules_d_pitch, unsigned rules_len, unsigned n)
{
    generate_random_normal_indices_kernel<<<(population_power / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
            rules_random_states_d, rules_random_states_d_pitch, population_power / 2, rules_len, n, indices_d);
    CUDA_CALL(hipDeviceSynchronize());
    perform_crossingover_kernel<<<population_power / 2, BLOCK_SIZE, sizeof(unsigned char[2][rules_len * (n+1)])>>>(
            indices_d, rules_d, rules_d_pitch, rules_len, n);
    CUDA_CALL(hipDeviceSynchronize());
}

extern "C"
void tune_lfs_gpu_impl(const unsigned *fsets_lens, unsigned rules_len, unsigned n, const GaussParams *xxs, const unsigned *ys, unsigned N, unsigned mu, unsigned lambda, unsigned it_number, GaussParams *gauss_params, unsigned char *rules)
{
    /*
     * GPU memory layout:
     * +==============================
     * | fsets_offsets, fsets_lens buffer
     * +------------------------------
     * | RandomState array [new_population_power, fsets_total_len]
     * +------------------------------
     * | GaussParams array [population_power, fsets_total_len]
     * +------------------------------
     * | NEW GaussParams array [new_population_power, fsets_total_len]
     * +------------------------------
     * | EvolutionaryParams array [population_power, fsets_total_len]
     * +------------------------------
     * | NEW EvolutionaryParams array [new_population_power, fsets_total_len]
     * +==============================
     * | RandomState array [new_population_power, rules_len]
     * +------------------------------
     * | rules array [population_power, rules_len, n+1]
     * +------------------------------
     * | NEW rules array [new_population_power, rules_len, n+1]
     * +==============================
     * | Input array [N, n]
     * +------------------------------
     * | Output buffer [N]
     * +==============================
     * | Union[scores, ps] buffer [new_population_power]
     * +------------------------------
     * | Indices buffer [new_population_power]
     * +==============================
     */

    unsigned population_power = mu;
    unsigned new_population_power = lambda;

    assert(new_population_power >= population_power);

    size_t i, it;
    size_t fsets_total_len = 0;

    unsigned fsets_offsets[n+1];

    {
        unsigned offset = 0;
        for (i = 0; i < n+1; ++i) {
            fsets_offsets[i] = offset;
            offset += fsets_lens[i];
            fsets_total_len += fsets_lens[i];
        }
    }

    size_t params_random_states_d_pitch, rules_random_states_d_pitch;
    size_t gauss_params_d_pitch, evolutionary_params_d_pitch;
    size_t rules_d_pitch, xxs_d_pitch;

    unsigned *fsets_offsets_d=0, *fsets_lens_d, *indices_d;
    RandomState *params_random_states_d, *rules_random_states_d;
    GaussParams *gauss_params_d, *new_gauss_params_d, *xxs_d;
    EvolutionaryParams *evolutionary_params_d, *new_evolutionary_params_d;
    unsigned char *rules_d, *new_rules_d;
    unsigned *ys_d;
    float scores[new_population_power], *scores_d;

    CUDA_CALL(hipMalloc(&fsets_offsets_d, sizeof(unsigned[n+1])));
    CUDA_CALL(hipMalloc(&fsets_lens_d, sizeof(unsigned[n+1])));
    CUDA_CALL(hipMallocPitch(&params_random_states_d, &params_random_states_d_pitch, sizeof(RandomState[fsets_total_len]), new_population_power));
    CUDA_CALL(hipMallocPitch(&gauss_params_d, &gauss_params_d_pitch, sizeof(GaussParams[fsets_total_len]), population_power));
    CUDA_CALL(hipMallocPitch(&new_gauss_params_d, &gauss_params_d_pitch, sizeof(GaussParams[fsets_total_len]), new_population_power));
    CUDA_CALL(hipMallocPitch(&evolutionary_params_d, &evolutionary_params_d_pitch, sizeof(EvolutionaryParams[fsets_total_len]), population_power));
    CUDA_CALL(hipMallocPitch(&new_evolutionary_params_d, &evolutionary_params_d_pitch, sizeof(EvolutionaryParams[fsets_total_len]), new_population_power));
    CUDA_CALL(hipMallocPitch(&rules_random_states_d, &rules_random_states_d_pitch, sizeof(RandomState[rules_len]), new_population_power));
    CUDA_CALL(hipMallocPitch(&rules_d, &rules_d_pitch, sizeof(unsigned char[rules_len][n+1]), population_power));
    CUDA_CALL(hipMallocPitch(&new_rules_d, &rules_d_pitch, sizeof(unsigned char[rules_len][n+1]), new_population_power));

    initialize_random_states(
            params_random_states_d, params_random_states_d_pitch, rules_random_states_d, rules_random_states_d_pitch,
            new_population_power, fsets_total_len, rules_len);

    DataBounds data_bounds[n+1];

    compute_data_bounds(xxs, ys, data_bounds, N, n);
    CUDA_CALL(hipMemcpy(fsets_offsets_d, fsets_offsets, sizeof(unsigned[n+1]), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(fsets_lens_d, fsets_lens, sizeof(unsigned[n+1]), hipMemcpyHostToDevice));

    initialize_population(
            params_random_states_d, params_random_states_d_pitch, rules_random_states_d, rules_random_states_d_pitch,
            data_bounds, fsets_lens, fsets_lens_d, fsets_total_len, population_power,
            gauss_params_d, gauss_params_d_pitch, evolutionary_params_d, evolutionary_params_d_pitch,
            rules_d, rules_d_pitch, rules_len, n, 0.001);

    CUDA_CALL(hipMallocPitch(&xxs_d, &xxs_d_pitch, sizeof(GaussParams[n]), N));
    CUDA_CALL(hipMalloc(&ys_d, sizeof(unsigned[N])));
    CUDA_CALL(hipMalloc(&scores_d, sizeof(float[new_population_power])));
    CUDA_CALL(hipMalloc(&indices_d, sizeof(unsigned[new_population_power])));

    CUDA_CALL(hipMemcpy2D(xxs_d, xxs_d_pitch, xxs, sizeof(GaussParams[n]), sizeof(GaussParams[n]), N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(ys_d, ys, sizeof(unsigned[N]), hipMemcpyHostToDevice));

    FILE *f = fopen("report_gpu.txt", "w");
    for (it = 0; it < it_number; ++it) {
        perform_reproduction(
                params_random_states_d, params_random_states_d_pitch, indices_d, population_power, new_population_power,
                new_gauss_params_d, gauss_params_d, gauss_params_d_pitch,
                new_evolutionary_params_d, evolutionary_params_d, evolutionary_params_d_pitch,
                new_rules_d, rules_d, rules_d_pitch, fsets_total_len, rules_len, n);

        // perform_crossingover(
        //         params_random_states_d, params_random_states_d_pitch, indices_d, new_population_power,
        //         new_rules_d, rules_d_pitch, rules_len, n);

        perform_mutation(
                params_random_states_d, params_random_states_d_pitch, params_random_states_d, params_random_states_d_pitch,
                scores_d, new_population_power, fsets_lens_d, fsets_total_len,
                new_gauss_params_d, gauss_params_d_pitch, new_evolutionary_params_d, evolutionary_params_d_pitch,
                new_rules_d, rules_d_pitch, rules_len, n, 0.1f / (rules_len*(n+1)));

        compute_scores(
                new_population_power, fsets_offsets_d,
                new_gauss_params_d, gauss_params_d_pitch, new_rules_d, rules_d_pitch, rules_len, n,
                xxs_d, xxs_d_pitch, ys_d, N, scores_d);

        perform_selection(
                scores_d, indices_d, population_power, new_population_power,
                gauss_params_d, new_gauss_params_d, gauss_params_d_pitch,
                evolutionary_params_d, new_evolutionary_params_d, evolutionary_params_d_pitch,
                rules_d, new_rules_d, rules_d_pitch, fsets_total_len, rules_len, n);

        // Debug purpose

        hipMemcpy(scores, scores_d, sizeof(float[population_power]), hipMemcpyDeviceToHost);

        {
            float avg = 0.f;
            printf("It [%3d] ", it);
            for (i = 0; i < population_power; ++i) {
                printf("%f ", scores[i]);
                avg += scores[i];
            }
            avg /= population_power;
            printf("Avg: %f\n", avg);
            fprintf(f, "%f\n", avg);
        }
    }
    fclose(f);

    hipMemcpy(gauss_params, gauss_params_d, sizeof(GaussParams[fsets_total_len]), hipMemcpyDeviceToHost);
    hipMemcpy(rules, rules_d, sizeof(unsigned char[rules_len][n]), hipMemcpyDeviceToHost);

    hipFree(indices_d);
    hipFree(scores_d);
    hipFree(ys_d);
    hipFree(xxs_d);
    hipFree(new_rules_d);
    hipFree(rules_d);
    hipFree(new_evolutionary_params_d);
    hipFree(evolutionary_params_d);
    hipFree(new_gauss_params_d);
    hipFree(gauss_params_d);
    hipFree(params_random_states_d);
    hipFree(fsets_lens_d);
    hipFree(fsets_offsets_d);
    CUDA_CALL(hipDeviceSynchronize());
}
